
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void kernel( int *a )
{
   int ix = blockIdx.x*blockDim.x + threadIdx.x;
   int iy = blockIdx.y*blockDim.y + threadIdx.y;
   int idx = iy * blockDim.x * gridDim.x + ix;

   a[idx] = a[idx] + 1;
}

int main()
{
  int *host_array;
  int *dev_array;

  host_array = (int *) malloc(sizeof(int)*16);
  hipMalloc(&dev_array, sizeof(int)*16);
  hipMemset(dev_array, 0, 16);

  dim3 block(2,2);
  dim3 threadPerBlock(2,2);
  kernel<<<block, threadPerBlock>>>(dev_array);
  hipMemcpy(host_array, dev_array, sizeof(int)*16, hipMemcpyDeviceToHost);

  for(int i = 0; i < 16; i++) printf(" %d ", host_array[i]);
  printf("\n");

  free(host_array);
  hipFree(dev_array);
  hipDeviceReset();
  return 0;
}
