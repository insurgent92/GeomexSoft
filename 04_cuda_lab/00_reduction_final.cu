
#include <hip/hip_runtime.h>
#include <stdio.h>

template <unsigned int blockSize>
__device__ void warpReduce(volatile int* sdata, int tid) {
      if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
      if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
      if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
      if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
      if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
      if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce6(int *g_idata, int *g_odata, int n) {

      extern __shared__ int sdata[];
      // perform first level of reduction, reading from global memory, writing to shared memory
      unsigned int tid = threadIdx.x;
      unsigned int i = blockIdx.x*(blockSize*2) + threadIdx.x;
      unsigned int gridSize = blockSize*2*gridDim.x;
      sdata[tid] = 0;
      while (i < n) {
            sdata[tid] += g_idata[i] + g_idata[i+blockSize];
            i += gridSize;
      }
      __syncthreads();

      // do reduction in shared mem
      if (blockSize >= 512) {
            if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
      }
      if (blockSize >= 256) {
            if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
      }
      if (blockSize >= 128) {
            if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();
      }

      if (tid < 32) warpReduce<blockSize>(sdata, tid);

      // write result for this block to global mem
      if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}



int main(void)
{
  long int size = 1 << 26;
  long int s;
  int sizeByte = size*sizeof(int);
  int* h_data = (int*) malloc(sizeByte);

  for(int i = 0; i < size; i++) {
    // h_data[i] = rand() & 0xFF;
    h_data[i] = i % 10;
  }

  long long int sum = 0;
  for(int i = 0; i < size; i++) sum += h_data[i];
  printf("CPU results = %lld \n", sum);

  int* d_idata = NULL;
  int* d_odata = NULL;
  hipMalloc(&d_idata, sizeByte);
  hipMalloc(&d_odata, sizeByte);

  hipMemcpy(d_idata, h_data, sizeByte, hipMemcpyHostToDevice);

  s = size >>  6;
  int blocks = (s+512-1)/512;
  reduce6<512><<<blocks/2, 512, 512*sizeof(int)>>>(d_idata, d_odata, size);
  hipDeviceSynchronize();
  printf("The size of array is %ld and it is processed on # of Blocks: %d \n", size, blocks/2);
  s = blocks/2;
  blocks = (s+512-1)/512;
  reduce6<512><<<blocks/2, 512, 512*sizeof(int)>>>(d_odata, d_idata, s);
  hipDeviceSynchronize();

  hipMemcpy(h_data, d_idata, sizeof(int), hipMemcpyDeviceToHost);
  printf("GPU result = %d\n", h_data[0]);

  hipFree(d_idata);
  hipFree(d_odata);
  free(h_data);
}
